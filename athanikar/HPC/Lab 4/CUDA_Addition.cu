#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>

__global__ void vectorAdd(int *A, int *B, int *C, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    if (idx < N) {
        C[idx] = A[idx] + B[idx];  
    }
}

int main() {
    int N = 11;  
    size_t size = N * sizeof(int);  
    int *A, *B, *C, *d_A, *d_B, *d_C;

    A = (int*)malloc(size);
    B = (int*)malloc(size);
    C = (int*)malloc(size);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    srand(time(NULL));
    for (int i = 1; i < N; i++) {
        A[i] = rand() % 100;  
        B[i] = rand() % 100;  
    }

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    int threadsPerBlock = 256;  
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;  
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    std::cout << "Vector A: ";
    for (int i = 1; i < N; i++) {
        std::cout << A[i] << " ";  
    }
    std::cout << std::endl;

    std::cout << "Vector B: ";
    for (int i = 1; i < N; i++) {
        std::cout << B[i] << " ";  
    }
    std::cout << std::endl;

    std::cout << "Result Vector C: ";
    for (int i = 1; i < N; i++) {
        std::cout << C[i] << " ";  
    }
    std::cout << std::endl;

    std::cout << "Calculations of Matrix C: " << std::endl;
    for (int i = 1; i < N; i++) {
        std::cout << "C[" << i << "] = " << A[i] << " + " << B[i] << " = " << C[i] << std::endl;
    }

    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}
